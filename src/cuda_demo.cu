// #include <cuda_runtime.h>
// #include <iostream>
// #include <cmath>

// __global__ void kernelToMeasure(double *sum, int size) {
//     int index = threadIdx.x + blockIdx.x * blockDim.x;
//     int stride = blockDim.x * gridDim.x;

//     for (int i = index; i < size; i += stride) {
//         sum[i] = __sinf(i) * __cosf(i);
//     }
// }

// int main() {
//     const int size = 1000000;
//     double *sum;
//     double *d_sum;

//     // 分配主机内存
//     sum = (double*)malloc(size * sizeof(double));

//     // 分配设备内存
//     cudaMalloc(&d_sum, size * sizeof(double));

//     // 定义线程块大小和网格大小
//     int blockSize = 256;
//     int numBlocks = (size + blockSize - 1) / blockSize;

//     // 创建 CUDA 事件
//     cudaEvent_t start, stop;
//     cudaEventCreate(&start);
//     cudaEventCreate(&stop);

//     // 记录开始时间
//     cudaEventRecord(start);

//     // 调用核函数
//     kernelToMeasure<<<numBlocks, blockSize>>>(d_sum, size);

//     // 记录结束时间
//     cudaEventRecord(stop);

//     // 等待事件完成
//     cudaEventSynchronize(stop);

//     // 计算运行时间
//     float milliseconds = 0;
//     cudaEventElapsedTime(&milliseconds, start, stop);

//     // 将结果从设备内存复制回主机内存
//     cudaMemcpy(sum, d_sum, size * sizeof(double), cudaMemcpyDeviceToHost);

//     // 处理结果（例如，计算总和）
//     double totalSum = 0.0;
//     for (int i = 0; i < size; ++i) {
//         totalSum += sum[i];
//     }
//     std::cout << "Total Sum: " << totalSum << std::endl;
//     std::cout << "Elapsed Time: " << milliseconds << " ms" << std::endl;

//     // 释放 CUDA 事件
//     cudaEventDestroy(start);
//     cudaEventDestroy(stop);

//     // 释放内存
//     free(sum);
//     cudaFree(d_sum);

//     return 0;
// }



#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

__global__ void kernelToMeasure(float *sum, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < size; i += stride) {
        sum[i] = __sinf(i) * __cosf(i);
    }
}

int main() {
    const int size = 1000000;
    float *sum;
    float *d_sum;

    // 分配主机内存
    sum = (float*)malloc(size * sizeof(float));
    memset(sum, 0, size * sizeof(float)); // 初始化内存

    // 分配设备内存
    hipMalloc(&d_sum, size * sizeof(float));

    // 定义线程块大小和网格大小
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    // 创建 CUDA 事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 记录开始时间
    hipEventRecord(start);

    // 调用核函数
    kernelToMeasure<<<numBlocks, blockSize>>>(d_sum, size);

    // 记录结束时间
    hipEventRecord(stop);

    // 等待事件完成
    hipEventSynchronize(stop);

    // 计算运行时间
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // 将结果从设备内存复制回主机内存
    hipMemcpy(sum, d_sum, size * sizeof(float), hipMemcpyDeviceToHost);

    // 处理结果（例如，计算总和）
    float totalSum = 0.0;
    for (int i = 0; i < size; ++i) {
        totalSum += sum[i];
    }
    std::cout << "Total Sum: " << totalSum << std::endl;
    std::cout << "Elapsed Time: " << milliseconds << " ms" << std::endl;

    // 释放 CUDA 事件
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // 释放内存
    free(sum);
    hipFree(d_sum);

    return 0;
}
